#include "hip/hip_runtime.h"
/*
 MD.c - a simple molecular dynamics program for simulating real gas properties of Lennard-Jones particles.
 
 Copyright (C) 2016  Jonathan J. Foley IV, Chelsea Sweet, Oyewumi Akinfenwa
 
 This program is free software: you can redistribute it and/or modify
 it under the terms of the GNU General Public License as published by
 the Free Software Foundation, either version 3 of the License, or
 (at your option) any later version.
 
 This program is distributed in the hope that it will be useful,
 but WITHOUT ANY WARRANTY; without even the implied warranty of
 MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 GNU General Public License for more details.
 
 You should have received a copy of the GNU General Public License
 along with this program.  If not, see <http://www.gnu.org/licenses/>.
 
 Electronic Contact:  foleyj10@wpunj.edu
 Mail Contact:   Prof. Jonathan Foley
 Department of Chemistry, William Paterson University
 300 Pompton Road
 Wayne NJ 07470
 
 */
#include "MDCuda.h"

// --------------------------CUDA --------------------------

#define NUM_THREADS_PER_BLOCK 512

// --------------------------CUDA --------------------------

// Number of particles
int N = 5000;

// --------------------------CUDA --------------------------

__device__ int N_Cuda;
__device__ double PE_Cuda;

// --------------------------CUDA --------------------------

double PEG;
double NA = 6.022140857e23;
double kBSI = 1.38064852e-23;  // m^2*kg/(s^2*K)

//  Size of box, which will be specified in natural units
double L;

//  Initial Temperature in Natural Units
double Tinit;  //2;

//  Vectors!
const int MAXPART=5001;
//  Position
double* r = (double *) malloc(MAXPART*3*sizeof(double));
//  Velocity
double* v= (double *) malloc(MAXPART*3*sizeof(double));
//  Acceleration
double* a= (double *) malloc(MAXPART*3*sizeof(double));

// --------------------------CUDA --------------------------

double *r_Cuda, *a_Cuda; 
double aux = N * 3 * sizeof(double);

// --------------------------CUDA --------------------------

char *atype = (char *)malloc(3 * sizeof(char));

//  Function prototypes

//  initialize positions on simple cubic lattice, also calls function to initialize velocities
void initialize();  

//  update positions and velocities using Velocity Verlet algorithm 
//  print particle coordinates to file for rendering via VMD or other animation software
//  return 'instantaneous pressure'
double VelocityVerlet(double dt, FILE *fp);  

//  Compute Force using F = -dV/dr
//  solve F = ma for use in Velocity Verlet
//  Compute total potential energy from particle coordinates
__global__ void computeAccelerationsGPU(double *a_Cuda, double *r_Cuda, double *Pot_Cuda);

//  Numerical Recipes function for generation gaussian distribution
double gaussdist();

//  Initialize velocities according to user-supplied initial Temperature (Tinit)
void initializeVelocities();

//  Compute mean squared velocity from particle velocities and total kinetic energy from particle mass and velocities
double MeanSquaredVelocityKin();


void computeAccelerations();

int main(){

    int i, NumTime;
    double dt, Vol, Temp, Press, Pavg = 0, Tavg = 0, rho, VolFac, TempFac, PressFac, timefac, KE, mvs, gc, Z;
    char prefix[1000], tfn[1000], ofn[1000], afn[1000];
    FILE *tfp, *ofp, *afp;
    
    scanf("%s",prefix);
    strcpy(tfn,prefix);
    strcat(tfn,"_traj.xyz");
    strcpy(ofn,prefix);
    strcat(ofn,"_output.txt");
    strcpy(afn,prefix);
    strcat(afn,"_average.txt");
    
    scanf("%s",atype);
    
    if (strcmp(atype,"He")==0) {
        
        VolFac = 1.8399744000000005e-29;
        PressFac = 8152287.336171632;
        TempFac = 10.864459551225972;
        timefac = 1.7572698825166272e-12;
        
    }
    else if (strcmp(atype,"Ne")==0) {
        
        VolFac = 2.0570823999999997e-29;
        PressFac = 27223022.27659913;
        TempFac = 40.560648991243625;
        timefac = 2.1192341945685407e-12;
        
    }
    else if (strcmp(atype,"Ar")==0) {
        
        VolFac = 3.7949992920124995e-29;
        PressFac = 51695201.06691862;
        TempFac = 142.0950000000000;
        timefac = 2.09618e-12;
        
    }
    else if (strcmp(atype,"Kr")==0) {
        
        VolFac = 4.5882712000000004e-29;
        PressFac = 59935428.40275003;
        TempFac = 199.1817584391428;
        timefac = 8.051563913585078e-13;
        
    }
    else if (strcmp(atype,"Xe")==0) {
        
        VolFac = 5.4872e-29;
        PressFac = 70527773.72794868;
        TempFac = 280.30305642163006;
        timefac = 9.018957925790732e-13;
        
    }
    else {
        
        VolFac = 3.7949992920124995e-29;
        PressFac = 51695201.06691862;
        TempFac = 142.0950000000000;
        timefac = 2.09618e-12;
        strcpy(atype,"Ar");
        
    }

    scanf("%lf",&Tinit);

    if (Tinit<0.) {
        printf("\n  !!!!! ABSOLUTE TEMPERATURE MUST BE A POSITIVE NUMBER!  PLEASE TRY AGAIN WITH A POSITIVE TEMPERATURE!!!\n");
        exit(0);
    }

    if (N>=MAXPART) {
        printf("\n\n\n  MAXIMUM NUMBER OF PARTICLES IS %i\n\n  PLEASE ADJUST YOUR INPUT FILE ACCORDINGLY \n\n", MAXPART);
        exit(0);
    }

    // Convert initial temperature from kelvin to natural units
    Tinit /= TempFac;
    
    scanf("%lf",&rho);

    // Copy N to the device variable N_Cuda
    hipMemcpyToSymbol(HIP_SYMBOL(N_Cuda), &N, sizeof(int));

    Vol = N/(rho*NA);
    
    Vol /= VolFac;

    if (Vol<N) {
        printf("\n\n\n  YOUR DENSITY IS VERY HIGH!\n\n");
        printf("  THE NUMBER OF PARTICLES IS %i AND THE AVAILABLE VOLUME IS %f NATURAL UNITS\n",N,Vol);
        printf("  SIMULATIONS WITH DENSITY GREATER THAN 1 PARTCICLE/(1 Natural Unit of Volume) MAY DIVERGE\n");
        printf("  PLEASE ADJUST YOUR INPUT FILE ACCORDINGLY AND RETRY\n\n");
        exit(0);
    }

    // Length of the box in natural units:
    L = cbrt(Vol);
    
    //  Files that we can write different quantities to
    tfp = fopen(tfn,"w");    //  The MD trajectory, coordinates of every particle at each timestep
    ofp = fopen(ofn,"w");    //  Output of other quantities (T, P, gc, etc) at every timestep
    afp = fopen(afn,"w");    //  Average T, P, gc, etc from the simulation
    
    NumTime = 200;
    dt = 0.5e-14/timefac;

    if (strcmp(atype,"He")==0) {
        dt = 0.2e-14/timefac;
        NumTime=50000;
    }

    initialize();
    
    computeAccelerations();


    fprintf(tfp,"%i\n",N);
    fprintf(ofp,"  time (s)              T(t) (K)              P(t) (Pa)           Kinetic En. (n.u.)     Potential En. (n.u.) Total En. (n.u.)\n");

    for (i=0; i<NumTime+1; i++) {
        
        Press = VelocityVerlet(dt, tfp);
        Press *= PressFac;

        mvs = MeanSquaredVelocityKin()/N;
        KE = MeanSquaredVelocityKin()*0.5;

        Temp = mvs/3 * TempFac;

        gc = NA*Press*(Vol*VolFac)/(N*Temp);
        Z  = Press*(Vol*VolFac)/(N*kBSI*Temp);
        
        Tavg += Temp;
        Pavg += Press;
         fprintf(ofp,"  %8.4e  %20.8f  %20.8f %20.8f  %20.8f  %20.8f \n",i*dt*timefac,Temp,Press,KE, PEG, KE+PEG);
        //fprintf(ofp,"  %12.8e  %12.8f  %12.8f %12.8f  %12.8f  %12.8f \n",i*dt*timefac,Temp,Press,KE, PEG, KE+PEG);
    }
    
    // Because we have calculated the instantaneous temperature and pressure,
    // we can take the average over the whole simulation here
    Pavg /= NumTime;
    Tavg /= NumTime;
    Z = Pavg*(Vol*VolFac)/(N*kBSI*Tavg);
    gc = NA*Pavg*(Vol*VolFac)/(N*Tavg);

/*
    fprintf(afp,"  Total Time (s)      T (K)               P (Pa)      PV/nT (J/(mol K))         Z           V (m^3)              N\n");
    fprintf(afp," --------------   -----------        ---------------   --------------   ---------------   ------------   -----------\n");
    fprintf(afp,"  %12.12e  %12.12f       %12.12f     %12.12f       %12.12f        %12.12e         %i\n",i*dt*timefac,Tavg,Pavg,gc,Z,Vol*VolFac,N);
  */  

    fprintf(afp,"  Total Time (s)      T (K)               P (Pa)      PV/nT (J/(mol K))         Z           V (m^3)              N\n");
    fprintf(afp," --------------   -----------        ---------------   --------------   ---------------   ------------   -----------\n");
    fprintf(afp,"  %8.4e  %15.5f       %15.5f     %10.5f       %10.5f        %10.5e         %i\n",i*dt*timefac,Tavg,Pavg,gc,Z,Vol*VolFac,N);
    
    printf("\n  AVERAGE TEMPERATURE (K):                 %15.5f\n",Tavg);
    printf("\n  AVERAGE PRESSURE  (Pa):                  %15.5f\n",Pavg);
    printf("\n  PV/nT (J * mol^-1 K^-1):                 %15.5f\n",gc);
    printf("\n  PERCENT ERROR of pV/nT AND GAS CONSTANT: %15.5f\n",100*fabs(gc-8.3144598)/8.3144598);
    printf("\n  THE COMPRESSIBILITY (unitless):          %15.5f \n",Z);
    printf("\n  TOTAL VOLUME (m^3):                      %10.5e \n",Vol*VolFac);
    printf("\n  NUMBER OF PARTICLES (unitless):          %i \n", N);
    
    free(r);
    free(v);
    free(a);
    free(atype);
    fclose(tfp);
    fclose(ofp);
    fclose(afp);
    
    return 0;
}

void initialize() {
    int n, p=0 , i, j, k;
    double pos,pos1, posm, pos2;
    
    // Number of atoms in each direction
    n = int(ceil(cbrt(N)));
    
    //  spacing between atoms along a given direction
    pos = L / n;
    posm = pos*0.5;
    
    //  index for number of particles assigned positions

    //  initialize positions
    for (i=0; i<n; i++) {
        pos1 = posm+i*pos ;
        for (j=0; j<n; j++) {
            pos2 = j*pos + posm;
            for (k=0; k<n; k++) {
                if (p<N*3) {
                    r[p++] = pos1;
                    r[p++] = pos2;
                    r[p++] = k*pos + posm;
                }
            }
        }
    }
    
    // Call function to initialize velocities
    initializeVelocities();
}

//  Function to calculate the averaged velocity squared
double MeanSquaredVelocityKin() { 
    
    double vaux = 0;
    
    for (int i=0; i<N*3; i++) {
        vaux += v[i]*v[i];
    }
    
    return vaux;
}

// --------------------------CUDA --------------------------



//   Uses the derivative of the Lennard-Jones potential to calculate
//   the forces on each atom.  Then uses a = F/m to calculate the
//   accelleration of each atom. 
__global__ void computeAccelerationsGPU(double *a_Cuda, double *r_Cuda, double *Pot_Cuda) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    __shared__ double sharedRk[NUM_THREADS_PER_BLOCK * 3];

    // Each thread loads the values of rk into shared memory
    for (int k = 0; k < 3; ++k) {
        sharedRk[threadIdx.x * 3 + k] = r_Cuda[i * 3 + k];
    }

    if (i < N_Cuda) {
        double local_VPot = 0.0;
        double v_CudaAux[3] = {0.0, 0.0, 0.0};

        for (int j = 0; j < N_Cuda; j++) {
            if (i != j) {

                double vals[3];
                double rij[3];
                double rSqd = 0;

                rij[0] = sharedRk[threadIdx.x * 3] - r_Cuda[j * 3];
                rij[1] = sharedRk[threadIdx.x * 3 + 1] - r_Cuda[j * 3 + 1];
                rij[2] = sharedRk[threadIdx.x * 3 + 2] - r_Cuda[j * 3 + 2];

                rSqd = rij[0] * rij[0] + rij[1] * rij[1] + rij[2] * rij[2];

                double rSqd3 = rSqd*rSqd*rSqd;
                double rSqd6 = rSqd3*rSqd3;
                local_VPot+=((1-rSqd3)/(rSqd6));
            
                double f = ((48 - 24*rSqd3)/(rSqd6*rSqd));

                vals[0] = rij[0] * f; 
                vals[1] = rij[1] * f; 
                vals[2] = rij[2] * f;

                v_CudaAux[0] += vals[0];
                v_CudaAux[1] += vals[1];
                v_CudaAux[2] += vals[2];
            }
        }
        Pot_Cuda[i] = local_VPot;

        a_Cuda[i * 3] = v_CudaAux[0];
        a_Cuda[i * 3 + 1] = v_CudaAux[1];
        a_Cuda[i * 3 + 2] = v_CudaAux[2];
    
    }
}

void computeAccelerations() {

    double Pot = 0.;
    double v_Pot[N];
    double* Pot_Cuda;

    int siz = N*3; 

    for (int i = 0; i < siz; i++) { 
        a[i] = 0;
    }

    for (int i = 0; i < N; i++)
        v_Pot[i] = 0;

    hipMalloc((void**)&r_Cuda, aux);
    hipMalloc((void**)&a_Cuda, aux);
    hipMalloc((void**)&Pot_Cuda, N * sizeof(double));
    checkCUDAError("Mem Allocation");

    hipMemcpy(a_Cuda, a, aux, hipMemcpyHostToDevice);
    hipMemcpy(r_Cuda, r, aux, hipMemcpyHostToDevice);
    checkCUDAError("Memcpy Host -> Device");

    int bpg = (N + NUM_THREADS_PER_BLOCK - 1) / NUM_THREADS_PER_BLOCK;  // Arredondamento para cima

    computeAccelerationsGPU<<<bpg, NUM_THREADS_PER_BLOCK>>>(a_Cuda, r_Cuda, Pot_Cuda);
    hipDeviceSynchronize();
    checkCUDAError("Error in computeAccelerationsGPU");

    hipMemcpy(a, a_Cuda, aux, hipMemcpyDeviceToHost);


    hipMemcpy(v_Pot, Pot_Cuda, N * sizeof(double), hipMemcpyDeviceToHost);
    checkCUDAError("Memcpy Device -> Host");

    for (int i = 0; i < N; i++) {
        Pot += v_Pot[i];
    }

    hipFree(r_Cuda);
    hipFree(a_Cuda);
    hipFree(Pot_Cuda);
    checkCUDAError("Free Mem");

    PEG= Pot*4;
}

// --------------------------CUDA --------------------------

double VelocityVerlet(double dt, FILE *fp) {
    
    int i;
    double psum = 0., temp1, temp2, dt1 = 0.5 * dt;

    for (i=0; i<N*3; i += 2) {
        temp1 = a[i] * dt1;
        r[i] += (v[i] + temp1) * dt;
        v[i] += temp1;

        temp2 = a[i+1] * dt1;
        r[i+1] += (v[i+1] + temp2) * dt;
        v[i+1] += temp2;
    }

    computeAccelerations();

    for (i=0; i<N*3; i += 2) {
        v[i] += a[i] * dt1;
        v[i+1] += a[i+1] * dt1;
    }
    
    // Elastic walls
    for (i=0; i<N*3; i += 2) {
        if (r[i]<0. || r[i]>=L) {
            v[i] *=-1.;
            psum += fabs(v[i]);
        }

        if (r[i+1]<0. || r[i+1]>=L) {
            v[i+1] *=-1.;
            psum += fabs(v[i+1]);
        }
    }
    
    return psum/(3*L*L*dt);
}

void initializeVelocities() {
    
    int i,j;
    for ( i=0; i < N*3; i += 2) {
        v[i] = gaussdist();
        v[i+1] = gaussdist();
    }
    
    double vCM[3] = {0, 0, 0};

    for ( i=0; i<N; i++) {
        for ( j=0; j<3; j++) {
            vCM[j] += v[i*3+j];
        }
    }

    for (i=0; i<3; i++) vCM[i] /= N;
    
     for (i=0; i<N; i++) {
        for (j=0; j<3; j++) {
            v[i*3+j] -= vCM[j];
        }
    }

    double vSqdSum, lambda;
    vSqdSum=0.;
    for (int i = 0; i < N * 3; i += 5) {
        vSqdSum += v[i] * v[i] + v[i + 1] * v[i + 1] + v[i + 2] * v[i + 2] + v[i + 3] * v[i + 3] + v[i + 4] * v[i + 4];
    }
    
    lambda = sqrt( 3*(N-1)*Tinit/vSqdSum);
    
    for (int i=0; i<N*3; i +=2) {
        v[i] *= lambda;
        v[i+1] *= lambda;
    }
}

//  Numerical recipes Gaussian distribution number generator
double gaussdist() {
    static bool available = false;
    static double gset;
    double fac, rsq, v1, v2;
    if (!available) {
        do {
            v1 = 2.0 * rand() / double(RAND_MAX) - 1.0;
            v2 = 2.0 * rand() / double(RAND_MAX) - 1.0;
            rsq = v1 * v1 + v2 * v2;
        } while (rsq >= 1.0 || rsq == 0.0);
        
        fac = sqrt(-2.0 * log(rsq) / rsq);
        gset = v1 * fac;
        available = true;
        
        return v2*fac;
    } else {
        
        available = false;
        return gset;
        
    }
}